
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 5 // Length of the vectors

__global__ void vectorAddBlock(int *a, int *b, int *c) {
    int tid = threadIdx.x;
    c[tid] = a[tid] + b[tid];
}

__global__ void vectorAddThread(int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    c[tid] = a[tid] + b[tid];
}

int main() {
    int *h_a, *h_b, *h_c; // Host vectors
    int *d_a, *d_b, *d_c; // Device vectors

    // Allocate memory on the host
    h_a = (int*)malloc(N * sizeof(int));
    h_b = (int*)malloc(N * sizeof(int));
    h_c = (int*)malloc(N * sizeof(int));

    // Initialize host vectors
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = 2 * i;
    }

    // Allocate memory on the device
    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));

    // Copy host vectors to device
    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);

    // Part A: Block size as N
    vectorAddBlock<<<1, N>>>(d_a, d_b, d_c);

    // Part B: N threads
    int numBlocks = (N + 255) / 256; // Use enough blocks to cover N threads
    vectorAddThread<<<numBlocks, 256>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < N; i++) {
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }

    // Free memory on the device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free memory on the host
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}