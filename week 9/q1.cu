
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 4;  // Matrix size (N x N)
const int BLOCK_SIZE = 2;

__global__ void matrixMul(int* A, int* B, int* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    if (row < n && col < n) {
        for (int i = 0; i < n; i++) {
            sum += A[row * n + i] * B[i * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    int h_A[N][N], h_B[N][N], h_C[N][N];  // Host matrices
    int *d_A, *d_B, *d_C;  // Device matrices
    int matrix_size = N * N * sizeof(int);
    

    // Initialize matrices h_A and h_B
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            h_A[i][j] = i;
            h_B[i][j] = i + j;
        }
    }

    // Allocate memory on the device
    hipMalloc((void**)&d_A, matrix_size);
    hipMalloc((void**)&d_B, matrix_size);
    hipMalloc((void**)&d_C, matrix_size);

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, matrix_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matrix_size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimGrid(N / BLOCK_SIZE, N / BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    // Launch the kernel
    matrixMul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    // Copy the result matrix from device to host
    hipMemcpy(h_C, d_C, matrix_size, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            printf("%d ", h_C[i][j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory

    // Perform further processing with the result matrix h_C
    // ...

    return 0;
}

